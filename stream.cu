#include "hip/hip_runtime.h"
/* CC: nvcc -g */
/* lib_list: -lmpi */
/* run: mpirun -l -n 2 */

#include <mpi.h>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

__global__ void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(int argc, char** argv) {
    MPI_Init(&argc, &argv);
    int mpi_errno;
    int rank, size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);
    printf("Process %d / %d\n", rank, size);

    //hipSetDevice(rank % 2); // Use different GPUs for each rank

    hipStream_t stream;
    hipStreamCreate(&stream);


    int N = 1000000;
    float *x, *y, *d_x, *d_y;
    x = (float*)malloc(N*sizeof(float));
    y = (float*)malloc(N*sizeof(float));

    hipMalloc(&d_x, N*sizeof(float)); 
    hipMalloc(&d_y, N*sizeof(float));

    if (rank == 0) {
        for (int i = 0; i < N; i++) {
            x[i] = 1.0f;
        }
    } else if (rank == 1) {
        for (int i = 0; i < N; i++) {
            y[i] = 2.0f;
        }
    }

    char str_stream[20];
    snprintf(str_stream, 20, "%d", stream);

    MPI_Info info;
    MPI_Info_create(&info);
    MPI_Info_set(info, "type", "hipStream_t");
    //MPI_Info_set(info, "value", &stream);
    MPI_Info_set(info, "id", str_stream);
    MPIX_Info_set_hex(info, "value", (void*)&stream, sizeof(hipStream_t));

    MPIX_Stream mpi_stream;
    MPIX_Stream_create(info, &mpi_stream);

    MPI_Comm stream_comm;
    MPIX_Stream_comm_create(MPI_COMM_WORLD, mpi_stream, &stream_comm);

    double start_time, end_time;
    MPI_Barrier(MPI_COMM_WORLD);
    start_time = MPI_Wtime();

#if 1    
    if (rank == 0) {
      #if 1  
        hipMemcpyAsync(d_x, x, N*sizeof(float), hipMemcpyHostToDevice, stream);

        mpi_errno = MPIX_Send_enqueue(d_x, N, MPI_FLOAT, 1, 0, stream_comm);
      #else  
        mpi_errno = MPIX_Send_enqueue(x, N, MPI_FLOAT, 1, 0, stream_comm);
      #endif  
        assert(mpi_errno == MPI_SUCCESS);

        hipStreamSynchronize(stream);
    } else if (rank == 1) {
        hipMemcpyAsync(d_y, y, N*sizeof(float), hipMemcpyHostToDevice, stream);

        mpi_errno = MPIX_Recv_enqueue(d_x, N, MPI_FLOAT, 0, 0, stream_comm, MPI_STATUS_IGNORE);
        assert(mpi_errno == MPI_SUCCESS);

        // Perform SAXPY on 1M elements
        saxpy<<<(N+255)/256, 256, 0, stream>>>(N, 2.0f, d_x, d_y);

        hipMemcpyAsync(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost, stream);

        hipStreamSynchronize(stream);
    }

#else
    if (rank == 0) {
        hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
        mpi_errno = MPI_Send(d_x, N, MPI_FLOAT, 1, 0, MPI_COMM_WORLD);
        assert(mpi_errno == MPI_SUCCESS);
    } else if (rank == 1) {
        hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
        mpi_errno = MPI_Recv(d_x, N, MPI_FLOAT, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        assert(mpi_errno == MPI_SUCCESS);

        // Perform SAXPY on 1M elements
        saxpy<<<(N+255)/256, 256, 0>>>(N, 2.0f, d_x, d_y);

        hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
    }

#endif
    end_time = MPI_Wtime();

    if (rank == 1) {
        float maxError = 0.0f;
        int errs = 0;
        for (int i = 0; i < N; i++) {
            if (abs(y[i] - 4.0f) > 0.01) {
                errs++;
                maxError = max(maxError, abs(y[i]-4.0f));
            }
        }
        printf("%d errors, Max error: %f\n", errs, maxError);
    }
    printf("Process %d finished in %f seconds\n", rank, end_time - start_time);

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);

    hipStreamDestroy(stream);
    MPI_Finalize();
}
